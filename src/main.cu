#include "hip/hip_runtime.h"
#ifndef MAX_THREADS
#define MAX_THREADS 1024
#endif
#ifndef MAX_BLOCKS
#define MAX_BLOCKS 65535
#endif

#include "../inc/hip/hip_runtime_api.h"
#include "clock.h"
#include "parse.h"
#include "plot.h"

#ifndef THRUST
#include "wos_kernel.cuh"
#endif

#include <limits>
#include <math_functions.h>

//#include <hipblas.h>
#ifdef THRUST
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>

// source:
// http://stackoverflow.com/questions/12614164/generating-a-random-number-vector-between-0-and-1-0-using-thrust

template <typename T>
struct prg {
  T a, b;
  __host__ __device__ prg(T _a = 0.f, T _b = 1.f) : a(_a), b(_b){};

  __host__ __device__ T operator()(unsigned int thread_id) const {
    thrust::default_random_engine rng;
    thrust::normal_distribution<T> dist(a, b);
    rng.discard(thread_id);

    return dist(rng);
  }
};

template <typename T>
struct getBoundaryDistance {
  T width;
  getBoundaryDistance(T _width) { width = _width; }

  __host__ __device__ T operator()(T &radius) const {
    return (1.0 - abs(radius));
  }
};
#endif

// initialize h_x0 vector of size dim and fill with val
template <typename T>
void initX0(T *x0, size_t dim, size_t len, T val);

int main(int argc, char *argv[]) {
  printTitle();
  printInfo("initializing");

// TODO: call WoS template wraper function

#ifdef DOUBLE
  typedef double T; // Type for problem
#else
  typedef float T;
#endif // DOUBLE
  Parameters p;

  // TODO this should/could go in parameter constructor
  int parseStatus = parseParams(argc, argv, p);
  if (parseStatus == 0)
    return 0;

  // TODO: Question: what effect does the d_eps have on practical convergence?
  T d_eps = 0.01; // 1 / sqrt(p.wos.x0.dimension); // or 0.01

  // instantiate timers
  Timers timers;

  timers.totalTimer.start();
#ifndef THRUST
  // declare local array variabls
  T *h_x0;
  checkCudaErrors(hipHostMalloc((void **)&h_x0, sizeof(T) * p.wos.x0.length));
  // declare pointers for device variables
  T *d_x0 = NULL;
  T *d_paths = NULL;
  // init our point on host
  // cast to T hotfix until class is templated
  initX0(h_x0, p.wos.x0.dimension, p.wos.x0.length, (T)p.wos.x0.value);

  timers.memorySetupTimer.start();

  // maloc device memory
  checkCudaErrors(hipMalloc((void **)&d_x0, p.wos.x0.length * sizeof(T)));

  printInfo("initializing d_paths");

  checkCudaErrors(hipMalloc((void **)&d_paths, p.wos.totalPaths * sizeof(T)));

  checkCudaErrors(hipMemset(d_paths, 0.0, p.wos.totalPaths * sizeof(T)));

  // Let's bring our data to the Device
  checkCudaErrors(hipMemcpy(d_x0, h_x0, p.wos.x0.length * sizeof(T),
                             hipMemcpyHostToDevice));

  timers.memorySetupTimer.end();
  timers.computationTimer.start();

  // Calling WoS kernel
  wos<T>(p, d_x0, d_paths, d_eps);
  hipDeviceSynchronize();
  timers.computationTimer.end();

  // We don't need d_x0 anymore, only to reduce solution data
  hipFree(d_x0);

#endif

#ifdef THRUST
  timers.memorySetupTimer.start();
  thrust::host_vector<T> h_x0(p.wos.x0.dimension);
  thrust::device_vector<T> d_x(p.wos.x0.dimension);
  thrust::device_vector<T> d_x0(p.wos.x0.dimension);
  thrust::fill_n(d_x0.begin(), p.wos.x0.dimension, (T)p.wos.x0.value);
  thrust::device_vector<T> d_radius(p.wos.x0.dimension);
  thrust::fill(d_radius.begin(), d_radius.end(), INFINITY);
  thrust::device_vector<T> d_direction(p.wos.x0.dimension);
  thrust::fill(d_direction.begin(), d_direction.end(), 0.0);
  thrust::device_vector<T> d_paths(p.wos.totalPaths);
  thrust::fill(d_paths.begin(), d_paths.end(), 0.0);
#ifdef PLOT
  thrust::device_vector<T> d_exitX(p.wos.totalPaths);
  thrust::fill(d_exitX.begin(), d_exitX.end(), 0.0);
  thrust::device_vector<T> d_exitY(p.wos.totalPaths);
  thrust::fill(d_exitY.begin(), d_exitY.end(), 0.0);
#endif // PLOT

  timers.memorySetupTimer.end();
  timers.computationTimer.start();
  T radius = INFINITY;
  T norm = 0.0;
  unsigned int position;
  T gpu_result = 0;
  T sum = 0.0;
  T squaredSum = 0.0;
  unsigned int counter = 0;
  unsigned int randCount = 0;

  thrust::counting_iterator<unsigned int> index_sequence_begin(0);
  for (unsigned int i = 0; i < p.wos.totalPaths; i++) {
    thrust::copy(d_x0.begin(), d_x0.end(), d_x.begin());

    radius = INFINITY;
    norm = 0.0;
    position = 0;
    counter = 0;
    while (d_eps <= radius) {
      // std::cout << " direction before rand:" << std::endl;
      // thrust::copy(d_direction.begin(), d_direction.end(),
      //              std::ostream_iterator<float>(std::cout, " "));
      // std::cout << "\n" << std::endl;
      // create random direction
      thrust::transform(index_sequence_begin + randCount,
                        index_sequence_begin + p.wos.x0.dimension + randCount,
                        d_direction.begin(), prg<T>(0.0, 1.0));
      // std::cout << " direction after rand:" << std::endl;
      // thrust::copy(d_direction.begin(), d_direction.end(),
      //              std::ostream_iterator<float>(std::cout, " "));
      // std::cout << "\n" << std::endl;

      sum += thrust::reduce(d_direction.begin(), d_direction.end());
      squaredSum += thrust::inner_product(
          d_direction.begin(), d_direction.end(), d_direction.begin(), (T)0.0);
      randCount += p.wos.x0.dimension;

      // normalize random direction
      // Source:
      // http://stackoverflow.com/questions/13688307/how-to-normalise-a-vector-with-thrust
      // std::cout << " direction before inner product:" << std::endl;
      // thrust::copy(d_direction.begin(), d_direction.end(),
      //              std::ostream_iterator<float>(std::cout, " "));
      // std::cout << "\n" << std::endl;
      norm = std::sqrt(thrust::inner_product(
          d_direction.begin(), d_direction.end(), d_direction.begin(), (T)0.0));
      // std::cout << " direction after inner_product:" << std::endl;
      // thrust::copy(d_direction.begin(), d_direction.end(),
      //              std::ostream_iterator<float>(std::cout, " "));
      // std::cout << "\n" << std::endl;

      using namespace thrust::placeholders;

      thrust::transform(d_direction.begin(), d_direction.end(),
                        d_direction.begin(), _1 / norm);

      thrust::transform(d_x.begin(), d_x.end(), d_radius.begin(),
                        getBoundaryDistance<T>((T)1.0));

      // calculate mimimun radius
      // Source:
      // http://stackoverflow.com/questions/7709181/finding-the-maximum-element-value-and-its-position-using-cuda-thrust
      thrust::device_vector<T>::iterator iter =
          thrust::min_element(d_radius.begin(), d_radius.end());

      radius = *iter;

      // std::cout << " before step:" << std::endl;
      // thrust::copy(d_x.begin(), d_x.end(),
      //              std::ostream_iterator<float>(std::cout, " "));
      // std::cout << "\n" << std::endl;

      // calculate next point X
      thrust::transform(d_direction.begin(), d_direction.end(), d_x.begin(),
                        d_x.end(), _2 += radius * _1);

      // std::cout << " after step:" << std::endl;
      // thrust::copy(d_x.begin(), d_x.end(),
      //              std::ostream_iterator<float>(std::cout, " "));
      // std::cout << "\n" << std::endl;

      counter++;
    }
    // std::cout << "while itterations: " << counter << std::endl;
    // Project current point to boundary
    thrust::transform(d_x.begin(), d_x.end(), d_radius.begin(),
                      getBoundaryDistance<T>((T)1.0));

    // find min element in radius
    thrust::device_vector<T>::iterator iter =
        thrust::min_element(d_radius.begin(), d_radius.end());

    position = iter - d_radius.begin();
    radius = *iter;

    // std::cout << " before projection:" << std::endl;
    // thrust::copy(d_x.begin(), d_x.end(),
    //              std::ostream_iterator<float>(std::cout, " "));
    // std::cout << "\n" << std::endl;

    // TODO: keep sign
    // project closest dimension to boundary
    thrust::fill(d_x.begin() + position, d_x.begin() + position + 1, (T)1.0);
#ifdef PLOT
    if (p.wos.x0.dimension == 2) {
      d_exitX[i] = d_x[0];
      d_exitY[i] = d_x[1];
    }
#endif
    // std::cout << "before boundary eval:" << std::endl;
    // thrust::copy(d_x.begin(), d_x.end(),
    //              std::ostream_iterator<float>(std::cout, " "));
    // std::cout << "\n" << std::endl;

    // evaluate boundary value
    d_paths[i] =
        thrust::inner_product(d_x.begin(), d_x.end(), d_x.begin(), (T)0.0) /
        (p.wos.x0.dimension * 2);

    // std::cout << "result vector in iteration " << i << " : " << std::endl;
    // thrust::copy(d_paths.begin(), d_paths.end(),
    //              std::ostream_iterator<float>(std::cout, " "));
    // std::cout << "\n" << std::endl;
  }
  std::cout << "mean: " << sum / randCount << std::endl;
  std::cout << "standard deviation: "
            << sqrt(pow(sum / randCount, 2) + squaredSum / randCount)
            << std::endl;
  timers.computationTimer.end();
#ifdef PLOT
  thrust::host_vector<T> h_paths(p.wos.totalPaths);
  thrust::host_vector<T> h_exitPoints(p.wos.x0.dimension * p.wos.totalPaths);
  thrust::host_vector<T> h_exitX(p.wos.totalPaths);
  thrust::host_vector<T> h_exitY(p.wos.totalPaths);
  thrust::copy(d_paths.begin(), d_paths.end(), h_paths.begin());
  thrust::copy(d_exitX.begin(), d_exitX.end(), h_exitX.begin());
  thrust::copy(d_exitY.begin(), d_exitY.end(), h_exitY.begin());

  exportData(h_paths.data(), h_exitX.data(), h_exitY.data(), p);

#endif
  gpu_result = thrust::reduce(thrust::device, d_paths.begin(), d_paths.end());
  gpu_result /= p.wos.totalPaths;
  timers.totalTimer.end();

#endif
#ifndef THRUST
#if defined(PLOT) || defined(CPU_REDUCE)
  printInfo("downloading path data");
  timers.memoryDownloadTimer.start();

  T h_paths[p.wos.totalPaths];
  // Download paths data
  checkCudaErrors(hipMemcpyAsync(
      &h_paths, d_paths, p.wos.totalPaths * sizeof(T), hipMemcpyDeviceToHost));

  timers.memoryDownloadTimer.end();
#endif

#ifdef PLOT
  plot(h_paths, p);
#endif

#ifdef CPU_REDUCE

  printInfo("reduce data on CPU");
  for (unsigned int i = 0; i < p.wos.totalPaths; i++) {
    std::cout << h_paths[i] << std::endl;
  }
  T gpu_result = reduceCPU(h_paths, p.wos.totalPaths);

#else

  T *h_results = (T *)malloc(p.reduction.blocks * sizeof(T));

  T *d_results;
  checkCudaErrors(
      hipMalloc((void **)&d_results, p.reduction.blocks * sizeof(T)));

  hipError_t err;
  reduce(p.wos.totalPaths, p.reduction.threads, p.reduction.blocks, d_paths,
         d_results);
  err = hipGetLastError();
  if (hipSuccess != err) {
    printf("Reduction Kernel returned an error:\n %s\n",
           hipGetErrorString(err));
  }

  timers.memoryDownloadTimer.start();

#ifdef DEBUG
  printf("[MAIN]: results values before copy:\n");
  for (int n = 0; n < p.reduction.blocks; n++) {
    printf("%f\n", h_results[n]);
  }
#endif

  // copy result from device to hostcudaStat =
  checkCudaErrors(hipMemcpy(h_results, d_results,
                             p.reduction.blocks * sizeof(T),
                             hipMemcpyDeviceToHost));

  timers.memoryDownloadTimer.end();

  T gpu_result = 0.0;
  for (int i = 0; i < p.reduction.blocks; i++) {
    printf("iteration %d, %f\n", i, h_results[i]);
    gpu_result += h_results[i];
  }
  free(h_results);
#endif

  gpu_result /= p.wos.totalPaths;

#ifdef DEBUG
  printf("[MAIN]: results values after copy:\n");
  for (int n = 0; n < p.reduction.blocks; n++) {
    printf("%f\n", h_results[n]);
  }
#endif

  timers.totalTimer.end();

  testResults((float)h_x0[0], (float)d_eps, (float)gpu_result, p);

  hipHostFree(h_x0);
#endif // !THRUST

#ifdef THRUST
  testResults((float)h_x0[0], (float)d_eps, gpu_result, p);
#endif // THRUST

  printTiming(timers.memorySetupTimer.get(), timers.computationTimer.get(),
              timers.totalTimer.get(), timers.memoryDownloadTimer.get());

#ifndef CPU_REDUCE
  hipFree(d_results);
#endif

  return (0);
}

template <typename T>
void initX0(T *h_x0, size_t dim, size_t len, T val) {
  // init our point on host
  for (unsigned int i = 0; i < dim; i++)
    // h_x0[i] = i == 1 ? 0.22 : 0;
    h_x0[i] = val;
  for (unsigned int i = dim; i < len; i++)
    h_x0[i] = 0.0;
}
