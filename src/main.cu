#include "hip/hip_runtime.h"
#include "../inc/hip/hip_runtime_api.h"
#include "clock.h"
#include "parse.h"
#include "plot.h"
#include "wos_kernel.cuh"

#include <limits>
#include <math_functions.h>

#ifndef MAX_THREADS
#define MAX_THREADS 1024
#endif
#ifndef MAX_BLOCKS
#define MAX_BLOCKS 65535
#endif
//#include <hipblas.h>

// initialize h_x0 vector of size dim and fill with val
template <typename T>
void initX0(T *x0, size_t dim, size_t len, T val);

int main(int argc, char *argv[]) {
  printTitle();
  printInfo("initializing");
  Parameters p;

  // TODO this should/could go in parameter constructor
  int parseStatus = parseParams(argc, argv, p);
  if (parseStatus == 0)
    return 0;

  // TODO: call WoS template wraper function
  // if (p.wos.typeDouble) {
  typedef double T; // Type for problem

  // } else {
  //   typedef float T; // Type for problem
  // }

  // TODO: Question: what effect does the d_eps have on practical convergence?
  T d_eps = 0.01; // 1 / sqrt(p.wos.x0.dimension); // or 0.01

  // instantiate timers
  Timers timers;

  timers.totalTimer.start();

  // declare local array variabls
  T *h_x0;
  checkCudaErrors(hipHostMalloc((void **)&h_x0, sizeof(T) * p.wos.x0.length));
  // declare pointers for device variables
  T *d_x0 = NULL;
  T *d_paths = NULL;
  // init our point on host
  // cast to T hotfix until class is templated
  initX0(h_x0, p.wos.x0.dimension, p.wos.x0.length, (T)p.wos.x0.value);

  timers.memorySetupTimer.start();

  // maloc device memory
  checkCudaErrors(hipMalloc((void **)&d_x0, p.wos.x0.length * sizeof(T)));

  printInfo("initializing d_paths");

  checkCudaErrors(hipMalloc((void **)&d_paths, p.wos.totalPaths * sizeof(T)));

  checkCudaErrors(hipMemset(d_paths, 0.0, p.wos.totalPaths * sizeof(T)));

  // Let's bring our data to the Device
  checkCudaErrors(hipMemcpy(d_x0, h_x0, p.wos.x0.length * sizeof(T),
                             hipMemcpyHostToDevice));

  timers.memorySetupTimer.end();
  timers.computationTimer.start();

  // Calling WoS kernel
  wos<T>(p, d_x0, d_paths, d_eps);
  hipDeviceSynchronize();
  timers.computationTimer.end();

  // We don't need d_x0 anymore, only to reduce solution data
  hipFree(d_x0);

#if defined(PLOT) || defined(CPU_REDUCE)
  printInfo("downloading path data\n");
  timers.memoryDownloadTimer.start();

  T h_paths[p.wos.totalPaths];
  // Download paths data
  checkCudaErrors(hipMemcpyAsync(
      &h_paths, d_paths, p.wos.totalPaths * sizeof(T), hipMemcpyDeviceToHost));

  timers.memoryDownloadTimer.end();
#endif

#ifdef PLOT
  plot(h_paths, p);
#endif

#ifdef CPU_REDUCE

  T gpu_result = reduceCPU(h_paths, p.wos.totalPaths);

#else

  T *h_results = (T *)malloc(p.reduction.blocks * sizeof(T));

  T *d_results;
  cudaCheckErrors(
      hipMalloc((void **)&d_results, p.reduction.blocks * sizeof(T)));

  hipError_t err;
  reduce(p.wos.totalPaths, p.reduction.threads, p.reduction.blocks, d_paths,
         d_results);
  err = hipGetLastError();
  if (hipSuccess != err) {
    printf("Reduction Kernel returned an error:\n %s\n",
           hipGetErrorString(err));
  }

  timers.memoryDownloadTimer.start();

#ifdef DEBUG
  printf("[MAIN]: results values before copy:\n");
  for (int n = 0; n < p.reduction.blocks; n++) {
    printf("%f\n", h_results[n]);
  }
#endif

  // copy result from device to hostcudaStat =
  cudaCheckErrors(hipMemcpy(h_results, d_results,
                             p.reduction.blocks * sizeof(T),
                             hipMemcpyDeviceToHost));

  timers.memoryDownloadTimer.end();

  T gpu_result = 0.0;
  for (int i = 0; i < p.reduction.blocks; i++) {
    printf("iteration %d, %f\n", i, h_results[i]);
    gpu_result += h_results[i];
  }
  free(h_results);
#endif

  gpu_result /= p.wos.totalPaths;

#ifdef DEBUG
  printf("[MAIN]: results values after copy:\n");
  for (int n = 0; n < p.reduction.blocks; n++) {
    printf("%f\n", h_results[n]);
  }
#endif

  timers.totalTimer.end();

  testResults((float)h_x0[0], (float)d_eps, (float)gpu_result, p);

  hipHostFree(h_x0);

  printTiming(timers.memorySetupTimer.get(), timers.computationTimer.get(),
              timers.totalTimer.get(), timers.memoryDownloadTimer.get());

#ifndef CPU_REDUCE
  hipFree(d_results);
#endif

  return (0);
}

template <typename T>
void initX0(T *h_x0, size_t dim, size_t len, T val) {
  // init our point on host
  for (unsigned int i = 0; i < dim; i++)
    // h_x0[i] = i == 1 ? 0.22 : 0;
    h_x0[i] = val;
  for (unsigned int i = dim; i < len; i++)
    h_x0[i] = 0.0;
}
